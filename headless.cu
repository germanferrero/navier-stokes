/*
  ======================================================================
   demo.c --- protoype to show off the simple solver
  ----------------------------------------------------------------------
   Author : Jos Stam (jstam@aw.sgi.com)
   Creation Date : Jan 9 2003

   Description:

	This code is a simple prototype that demonstrates how to use the
	code provided in my GDC2003 paper entitles "Real-Time Fluid Dynamics
	for Games". This code uses OpenGL and GLUT for graphics and interface

  =======================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include "cuda_helper.h"
#include "indices.h"
#include "wtime.h"
/* macros */
#include "solver.h"

#define IX(x,y) (rb_idx((x),(y),(N+2)))


/* global variables */

static int N;
static float dt, diff, visc;
static float force, source;

static float *u, *v, *u_prev, *v_prev;
static float *dens, *dens_prev;
static float *velocity2;


/*
  ----------------------------------------------------------------------
   free/clear/allocate simulation data
  ----------------------------------------------------------------------
*/


static void free_data(void)
{
    if (u) {
        checkCudaCall(hipFree(u));
    }
    if (v) {
        checkCudaCall(hipFree(v));
    }
    if (u_prev) {
        checkCudaCall(hipFree(u_prev));
    }
    if (v_prev) {
        checkCudaCall(hipFree(v_prev));
    }
    if (dens) {
        checkCudaCall(hipFree(dens));
    }
    if (dens_prev) {
        checkCudaCall(hipFree(dens_prev));
    }
    if (velocity2) {
        checkCudaCall(hipFree(velocity2));
    }
}

static void clear_data(void)
{
    int i, size = (N + 2) * (N + 2);
    for (i = 0; i < size; i++) {
        u[i] = v[i] = u_prev[i] = v_prev[i] = dens[i] = dens_prev[i] = velocity2[i] = 0.0f;
    }
}

static int allocate_data(void)
{
    int size = (N + 2) * (N + 2);

    checkCudaCall(hipMallocManaged(&u, size * sizeof(float)));
    checkCudaCall(hipMallocManaged(&v, size * sizeof(float)));
    checkCudaCall(hipMallocManaged(&u_prev, size * sizeof(float)));
    checkCudaCall(hipMallocManaged(&v_prev, size * sizeof(float)));
    checkCudaCall(hipMallocManaged(&dens, size * sizeof(float)));
    checkCudaCall(hipMallocManaged(&dens_prev, size * sizeof(float)));
    checkCudaCall(hipMallocManaged(&velocity2, size * sizeof(float)));

    return (1);
}

static void react(float* velocity2, float* d, float* u, float* v)
{
    int size = (N + 2) * (N + 2);
    float *max_velocity2;
    checkCudaCall(hipMallocManaged(&max_velocity2, sizeof(float)));
    float *max_density;
    checkCudaCall(hipMallocManaged(&max_density, sizeof(float)));

    launcher_get_velocity2(velocity2, N, u, v);
    void *v2_temp_storage = NULL;
    size_t v2_temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(v2_temp_storage, v2_temp_storage_bytes, velocity2, max_velocity2, size);
    checkCudaCall(hipMalloc(&v2_temp_storage, v2_temp_storage_bytes));
    hipcub::DeviceReduce::Max(v2_temp_storage, v2_temp_storage_bytes, velocity2, max_velocity2, size);

    void *d_temp_storage = NULL;
    size_t d_temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, d_temp_storage_bytes, d, max_density, size);
    checkCudaCall(hipMalloc(&d_temp_storage, d_temp_storage_bytes));
    hipcub::DeviceReduce::Max(d_temp_storage, d_temp_storage_bytes, d, max_density, size);

    checkCudaCall(hipMemset(u, 0, size * sizeof(float)));
    checkCudaCall(hipMemset(v, 0, size * sizeof(float)));
    checkCudaCall(hipMemset(d, 0, size * sizeof(float)));

    launcher_add_forces(max_velocity2, N, force, u, v);
    launcher_add_densities(max_density, N, source, d);

    return;
}

static void one_step(void)
{
    static int times = 1;
    static double start_t = 0.0;
    static double start_total_t = 0.0;
    static double total_cells_p_s = 0.0;
    static double one_second = 0.0;
    static double react_cells_p_s = 0.0;
    static double vel_cells_p_s = 0.0;
    static double dens_cells_p_s = 0.0;

    start_total_t = wtime();
    start_t = wtime();
    react(velocity2, dens_prev, u_prev, v_prev);
    react_cells_p_s += (N * N) / (wtime() - start_t);

    start_t = wtime();
    vel_step(N, u, v, u_prev, v_prev, visc, dt);
    vel_cells_p_s += (N * N) / (wtime() - start_t);

    start_t = wtime();
    dens_step(N, dens, dens_prev, u, v, diff, dt);
    dens_cells_p_s += (N * N) / (wtime() - start_t);

    total_cells_p_s += (N * N) / (wtime() - start_total_t);
    if (1.0 < wtime() - one_second) { /* at least 1s between stats */
        fprintf(stderr, "%lf, %lf, %lf, %lf: cells per second total step, react, vel_step, dens_step\n",
                total_cells_p_s / times,
                react_cells_p_s / times, vel_cells_p_s / times, dens_cells_p_s / times);
        one_second = wtime();
        react_cells_p_s = 0.0;
        vel_cells_p_s = 0.0;
        dens_cells_p_s = 0.0;
        total_cells_p_s = 0.0;
        times = 1;
    } else {
        times++;
    }
}


/*
  ----------------------------------------------------------------------
   main --- main routine
  ----------------------------------------------------------------------
*/

int main(int argc, char** argv)
{
    int i = 0;
    static double start_t = 0.0;
    static double end_t = 0.0;

    if (argc != 1 && argc != 7) {
        fprintf(stderr, "usage : %s N dt diff visc force source\n", argv[0]);
        fprintf(stderr, "where:\n");
        fprintf(stderr, "\t N      : grid resolution\n");
        fprintf(stderr, "\t dt     : time step\n");
        fprintf(stderr, "\t diff   : diffusion rate of the density\n");
        fprintf(stderr, "\t visc   : viscosity of the fluid\n");
        fprintf(stderr, "\t force  : scales the mouse movement that generate a force\n");
        fprintf(stderr, "\t source : amount of density that will be deposited\n");
        exit(1);
    }

    if (argc == 1) {
        N = 128;
        dt = 0.1f;
        diff = 0.0f;
        visc = 0.0f;
        force = 5.0f;
        source = 100.0f;
        fprintf(stderr, "Using defaults : N=%d dt=%g diff=%g visc=%g force = %g source=%g\n",
                N, dt, diff, visc, force, source);
    } else {
        N = atoi(argv[1]);
        dt = atof(argv[2]);
        diff = atof(argv[3]);
        visc = atof(argv[4]);
        force = atof(argv[5]);
        source = atof(argv[6]);
    }

    if (!allocate_data()) {
        exit(1);
    }
    clear_data();

    for (i = 0; i < 8; i++) {
        if (i == 2) {
            start_t = wtime();
        }
        one_step();
    }
    end_t = wtime();
    printf("%lf\n", ((N * N) * (8 - 2)) / (end_t - start_t));
    free_data();

    exit(0);
}
